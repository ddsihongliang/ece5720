#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

#define EPS2 1.0E-9

__device__ float3
bodyBodyInteraction(float4 bi, float4 bj, float3 ai)
{
  float3 r;

  // r_ij [3 FLOPS]
  r.x = bj.x - bi.x;
  r.y = bj.y - bi.y;
  r.z = bj.z - bi.z;
  // distSqr = dot(r_ij, r_ij) + EPS^2 [6 FLOPS]
  float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;

  // invDistCube =1/distSqr^(3/2) [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
  float distSixth = distSqr * distSqr * distSqr;
  float invDistCube = 1.0f/sqrtf(distSixth);

  // s = m_j * invDistCube [1 FLOP]
  float s = bj.w * invDistCube;

  //a_i= a_i+s*r_ij[6FLOPS]
  ai.x += r.x * s;
  ai.y += r.y * s;
  ai.z += r.z * s;
  //printf("ai.x : %f\n", ai.x);
  return ai;
}

__global__ void
calculate_forces(float4 *devX, float4 *devA, int N, int numTiles)
{
  extern __shared__ float4 shPosition[];

  float4 *globalX = devX;
  float4 *globalA = devA;
  float4 myPosition;
  float3 acc = {0.0f, 0.0f, 0.0f};
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gtid >= N) return;
  myPosition = globalX[gtid];

  for (int i = 0; i < numTiles; i++) {
    int idx = i * blockDim.x + threadIdx.x;
    shPosition[threadIdx.x] = globalX[idx];
    __syncthreads();
    //printf("myPosition.x = %f, shPosition[%d].x = %f, globalX[%d].x = %f\n", myPosition.x, threadIdx.x, shPosition[threadIdx.x].x, idx, globalX[idx].x);
    for (int j = 0; j < blockDim.x; j++) {
      acc = bodyBodyInteraction(myPosition, shPosition[j], acc);
    }
    __syncthreads();
  }

  // Save the result in global memory for the integration step.
  float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
  globalA[gtid] = acc4;
}
