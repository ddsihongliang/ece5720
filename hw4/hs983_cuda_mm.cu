/***********************************************************************
To Compile:
 /usr/local/cuda-10.0/bin/nvcc -arch=compute_52 -o file.out filename.cu
***********************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

void check_output(float *A, int dim) {
  /* Print output for debug */
  int i,j;
  printf("\n");
  for(i = 0; i < dim; i++) {
    for(j = 0; j < dim; j++) {
      printf("%3.4f ", A[i*dim + j]);
    }
    printf(";\n");
  }
  printf("\n");
}

__global__ void MyKernel(float *d_a,float *d_b,float *d_c,int dim){
//  __shared__ float shared[1024];
  float partial = 0.0;

  int i = threadIdx.y + blockIdx.y * blockDim.y; //row i of c
  int j = threadIdx.x + blockIdx.x * blockDim.x; //Column j of c
  int k;
  i = i*dim;

  for(k = 0; k < dim; k++){
    partial+=d_a[i+k] * d_b[k*dim+j];
  }
  d_c[i+j] = partial;
}

__global__ void MyKernel2(float *d_a,float *d_b,float *d_c,int dim){
  extern __shared__ float s[];  // declear a single shared array.
  float *a_tile = s;           // Divide the shared array into two
  float *b_tile = (float*)&a_tile[blockDim.x*blockDim.y];

  float partial = 0.0;
  int bx = blockIdx.x ; int by = blockIdx.y ;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int i = by * blockDim.y + ty; //row i of c
  int j = bx * blockDim.x + tx; //Column j of c
  int k,m;
  i = i * dim;
  int y = ty * blockDim.y;

  for(m = 0; m < dim/blockDim.x; m=m+blockDim.x) {
     a_tile[y+tx] = d_a[i + (m+tx)]; /* load coalesced */
     b_tile[y+tx] = d_b[(m+ty)*dim + j]; /* not coalesced */
    __syncthreads();
    for(k = 0; k < blockDim.x; ++k)
      partial += a_tile[y+k] * b_tile[k*blockDim.y+tx]; /* A bank conflicts */
    __syncthreads();
    d_c[i+j] = partial;
  }
}

__global__ void MyKernel3(float *d_a,float *d_b,float *d_cT,int dim){
  extern __shared__ float s[];  // declear a single shared array.
  float *a_tile = s;           // Divide the shared array into two
  float *bT_tile = (float*)&a_tile[blockDim.x*blockDim.y];

  float partial = 0.0;
  int bx = blockIdx.x ; int by = blockIdx.y ;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int i = by * blockDim.y + ty; //row i of c
  int j = bx * blockDim.x + tx; //Column j of c
  int k,m;
  i = i * dim;
  int y = ty * blockDim.y;

  for(m = 0; m < dim/blockDim.x; m=m+blockDim.x) {
      a_tile[y+tx] = d_a[i + (m+tx)]; /* load coalesced */
     bT_tile[y+tx] = d_b[i + (m+tx)]; /* load coalesced */
    __syncthreads();
    for(k = 0; k < blockDim.x; ++k)
      partial += a_tile[ty+k*blockDim.x]*bT_tile[tx+k*blockDim.y]; /* No bank conflicts */
    __syncthreads();
    d_cT[i+j] = partial;
  }
}

int main(int argc, char const *argv[]) {
  // Initiailize matrix dimension
  int dim = 1024,block_size = 32;
  int i, grid_size;
  if (argc > 1) {
    dim = atoi(argv[1]);
    block_size = atoi(argv[2]);
  }
  // declear host and device timer.
  srand(3);
  grid_size = dim / block_size;
  dim3 Block(block_size,block_size);
  dim3 Grid(grid_size,grid_size);
  struct timespec start,finish;
    int ntime, stime;
  float tot_time=0.0;

  // Populate matrice
  float *a  = (float*)malloc(sizeof(float)*dim*dim);
  float *bT = (float*)malloc(sizeof(float)*dim*dim);
  float *c  = (float*)malloc(sizeof(float)*dim*dim);
  float *d_a, *d_bT ,*d_c, limit=10.0; //d_bT for transposed

  for(i = 0; i < dim*dim; i++){
    a[i]  = ((float)rand()/(float)(RAND_MAX)) * limit;
    bT[i] = ((float)rand()/(float)(RAND_MAX)) * limit;
  }

  // Allocate device memeory.
  hipMalloc( (void**)&d_a,  dim*dim*sizeof(float));
  hipMalloc( (void**)&d_bT, dim*dim*sizeof(float));
  hipMalloc( (void**)&d_c,  dim*dim*sizeof(float));

  // Initiailize timer & start recording.
  clock_gettime(CLOCK_REALTIME, &start);

  // Copy memory to device.
  hipMemcpy(d_a ,a ,dim*dim*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_bT,bT,dim*dim*sizeof(float),hipMemcpyHostToDevice);

  // Call CUDA kernel function.
  MyKernel<<<Grid, Block>>>(d_a,d_bT,d_c,dim);
  hipMemcpy(c, d_c, sizeof(float)*dim*dim,hipMemcpyDeviceToHost);

  // Timer stop.
  hipDeviceSynchronize();
  clock_gettime(CLOCK_REALTIME, &finish);
  ntime = finish.tv_nsec - start.tv_nsec;
  stime = (int)finish.tv_sec - (int) start.tv_sec;
  tot_time = ntime*1.0E-9 + stime;

  /* Print output for debug */
  printf("kernel#1 Time elapsed: %f ms. matrix dimension: %d X %d\n",
  tot_time*1.0E3,dim,dim);

  // reset memory and timer.
  hipFree(d_c); hipFree(d_bT); hipFree(d_a);

  /*----------------Tile method with bank conflicts:------------------------*/
  // Allocate memory again:
  hipMalloc( (void**)&d_a,  dim*dim*sizeof(float));
  hipMalloc( (void**)&d_bT, dim*dim*sizeof(float));
  hipMalloc( (void**)&d_c,  dim*dim*sizeof(float));

  // start timming.
  clock_gettime(CLOCK_REALTIME, &start);

  hipMemcpy(d_a ,a ,dim*dim*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_bT,bT,dim*dim*sizeof(float),hipMemcpyHostToDevice);

  MyKernel2<<<Grid,Block,(2*Block.x*Block.y*sizeof(float))>>>(d_a,d_bT,d_c,dim);
  hipMemcpy(c, d_c, sizeof(float)*dim*dim,hipMemcpyDeviceToHost);

  // Timer stop.
  hipDeviceSynchronize();
  clock_gettime(CLOCK_REALTIME, &finish);
  ntime = finish.tv_nsec - start.tv_nsec;
  stime = (int)finish.tv_sec - (int) start.tv_sec;
  tot_time = ntime*1.0E-9 + stime;

  /* Print output for debug */
  printf("kernel#2 Time elapsed: %f ms. matrix dimension: %d X %d\n",
  tot_time*1.0E3,dim,dim);

  // reset memory and timer.
  hipFree(d_c); hipFree(d_bT); hipFree(d_a);

  /*----------------Tile method with no bank conflicts:----------------------*/
  // Allocate memory again:
  hipMalloc( (void**)&d_a,  dim*dim*sizeof(float));
  hipMalloc( (void**)&d_bT, dim*dim*sizeof(float));
  hipMalloc( (void**)&d_c,  dim*dim*sizeof(float));

  // start timming.
  clock_gettime(CLOCK_REALTIME, &start);

  hipMemcpy(d_a ,a ,dim*dim*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_bT,bT,dim*dim*sizeof(float),hipMemcpyHostToDevice);

  MyKernel3<<<Grid,Block,(2*Block.x*Block.y*sizeof(float))>>>(d_a,d_bT,d_c,dim);
  hipMemcpy(c, d_c, sizeof(float)*dim*dim,hipMemcpyDeviceToHost);

  // Timer stop.
  hipDeviceSynchronize();
  clock_gettime(CLOCK_REALTIME, &finish);
  ntime = finish.tv_nsec - start.tv_nsec;
  stime = (int)finish.tv_sec - (int) start.tv_sec;
  tot_time = ntime*1.0E-9 + stime;

  /* Print output for debug */
  printf("kernel#3 Time elapsed: %f ms. matrix dimension: %d X %d\n",
  tot_time*1.0E3,dim,dim);

  // reset memory and timer.
  hipFree(d_c); hipFree(d_bT); hipFree(d_a);
  free(a); free(bT); free(c);
  return 0;
}
